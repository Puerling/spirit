#include "hip/hip_runtime.h"
#include <Vectormath.hpp>
#include <Vectormath_Defines.hpp>

#include <Eigen/Core>
#include <Eigen/Dense>

namespace Vectormath
{

__global__ void cu_fill( scalar * sf, scalar s, const size_t N )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N )
    {
        sf[idx] = s;
    }
}
void fill( scalarfield & sf, scalar s )
{
    unsigned int n = sf.size();
    cu_fill<<<( n + 1023 ) / 1024, 1024>>>( sf.data(), s, n );
    CU_CHECK_AND_SYNC();
}
__global__ void cu_fill_mask( scalar * sf, scalar s, const int * mask, const size_t N )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N )
    {
        sf[idx] = mask[idx] * s;
    }
}
void fill( scalarfield & sf, scalar s, const intfield & mask )
{
    unsigned int n = sf.size();
    cu_fill_mask<<<( n + 1023 ) / 1024, 1024>>>( sf.data(), s, mask.data(), n );
    CU_CHECK_AND_SYNC();
}

__global__ void cu_fill( Vector3 * vf1, Vector3 v2, const size_t N )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N )
    {
        vf1[idx] = v2;
    }
}
void fill( vectorfield & vf, const Vector3 & v )
{
    unsigned int n = vf.size();
    cu_fill<<<( n + 1023 ) / 1024, 1024>>>( vf.data(), v, n );
    CU_CHECK_AND_SYNC();
}
__global__ void cu_fill_mask( Vector3 * vf1, Vector3 v2, const int * mask, const size_t N )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N )
    {
        vf1[idx] = v2;
    }
}
void fill( vectorfield & vf, const Vector3 & v, const intfield & mask )
{
    unsigned int n = vf.size();
    cu_fill_mask<<<( n + 1023 ) / 1024, 1024>>>( vf.data(), v, mask.data(), n );
    CU_CHECK_AND_SYNC();
}

} // namespace Vectormath
