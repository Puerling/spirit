#include "hip/hip_runtime.h"
#include <Indexing.hpp>
#include <Interaction.hpp>
#include <Vectormath_Defines.hpp>

#include <Eigen/Core>
#include <Eigen/Dense>

using Indexing::cu_check_atom_type;

__global__ void CU_E_Biaxial_Anisotropy(
    const Vector3 * spins, const int * atom_types, const int n_cell_atoms, const int n_anisotropies,
    const int * indices, const PolynomialBasis * bases, const unsigned int * site_p, const PolynomialTerm * terms,
    scalar * energy, size_t n_cells_total )
{
    for( auto icell = blockIdx.x * blockDim.x + threadIdx.x; icell < n_cells_total; icell += blockDim.x * gridDim.x )
    {
        for( int iani = 0; iani < n_anisotropies; ++iani )
        {
            int ispin = icell * n_cell_atoms + indices[iani];
            if( cu_check_atom_type( atom_types[ispin] ) )
            {
                const scalar s1 = bases[iani].k1.dot( spins[ispin] );
                const scalar s2 = bases[iani].k2.dot( spins[ispin] );
                const scalar s3 = bases[iani].k3.dot( spins[ispin] );

                const scalar sin_theta_2 = 1 - s1 * s1;

                scalar result = 0;
                for( int iterm = site_p[iani]; iterm < site_p[iani + 1]; ++iterm )
                {
                    const auto & [coeff, n1, n2, n3] = terms[iterm];
                    result += coeff * pow( sin_theta_2, n1 ) * pow( s2, n2 ) * pow( s3, n3 );
                }
                energy[ispin] += result;
            }
        }
    }
}

void Interaction::Energy_per_Spin( const Geometry & geometry, const vectorfield & spins, scalarfield & energy )
{
    const int size = geometry.n_cells_total;
    CU_E_Biaxial_Anisotropy<<<( size + 1023 ) / 1024, 1024>>>(
        spins.data(), geometry.atom_types.data(), geometry.n_cell_atoms, this->indices.size(), this->indices.data(),
        this->bases.data(), this->site_p.data(), this->terms.data(), energy.data(), size );
    CU_CHECK_AND_SYNC();
}

__global__ void CU_Gradient_Biaxial_Anisotropy(
    const Vector3 * spins, const int * atom_types, const int n_cell_atoms, const int n_anisotropies,
    const int * indices, const PolynomialBasis * bases, const unsigned int * site_p, const PolynomialTerm * terms,
    Vector3 * gradient, size_t n_cells_total )
{
    for( auto icell = blockIdx.x * blockDim.x + threadIdx.x; icell < n_cells_total; icell += blockDim.x * gridDim.x )
    {
        for( int iani = 0; iani < n_anisotropies; ++iani )
        {
            int ispin = icell * n_cell_atoms + indices[iani];
            if( cu_check_atom_type( atom_types[ispin] ) )
            {
                Vector3 result = Vector3::Zero();

                const auto & [k1, k2, k3] = bases[iani];

                const scalar s1 = k1.dot( spins[ispin] );
                const scalar s2 = k2.dot( spins[ispin] );
                const scalar s3 = k3.dot( spins[ispin] );

                const scalar sin_theta_2 = 1 - s1 * s1;

                for( auto iterm = site_p[iani]; iterm < site_p[iani + 1]; ++iterm )
                {
                    const auto & [coeff, n1, n2, n3] = terms[iterm];

                    const scalar a = pow( s2, n2 );
                    const scalar b = pow( s3, n3 );
                    const scalar c = pow( sin_theta_2, n1 );

                    if( n1 > 0 )
                        result += k1 * ( coeff * a * b * n1 * ( -2.0 * s1 * pow( sin_theta_2, n1 - 1 ) ) );
                    if( n2 > 0 )
                        result += k2 * ( coeff * b * c * n2 * pow( s2, n2 - 1 ) );
                    if( n3 > 0 )
                        result += k3 * ( coeff * a * c * n3 * pow( s3, n3 - 1 ) );
                }

                gradient[ispin] += result;
            }
        }
    }
}

void Interaction::Gradient( const Geometry & geometry, const vectorfield & spins, vectorfield & gradient )
{
    const int size = geometry.n_cells_total;
    CU_Gradient_Biaxial_Anisotropy<<<( size + 1023 ) / 1024, 1024>>>(
        spins.data(), geometry.atom_types.data(), geometry.n_cell_atoms, this->indices.size(), this->indices.data(),
        this->bases.data(), this->site_p.data(), this->terms.data(), gradient.data(), size );
    CU_CHECK_AND_SYNC();
};
